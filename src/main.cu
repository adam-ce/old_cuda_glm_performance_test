#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2014 Adam Celarek
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
*/

#include <random>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>
#include "hip/hip_vector_types.h"
#include <stdio.h>
#include <chrono>
#include <glm/gtx/matrix_major_storage.hpp>

#define NUM_ELEMENTS 2000000
#define THREADS_PER_BLOCK 256

//handle cuda errors
void hce(hipError_t error)
{
    if(error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }
}

__global__ void cuMatrixKernel(const float4 *vectors, mat4 matrix, float4 *result, int numElements, int innerLoopSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements) {
        result[i] = matrix * vectors[i];
        if(i > 3) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i] = matrix * result[i];
                result[i] += matrix * vectors[i-1];
                result[i] += matrix * vectors[i-2];
                result[i] += matrix * vectors[i-3];
                result[i] += matrix * vectors[i-4];
            }
        }
    }
}
__global__ void cuDotKernel(const float4 *vectors, float4 *result, int numElements, int innerLoopSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements) {
        result[i] = vectors[i];
        if(i>1 && i < NUM_ELEMENTS - 1) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i].y = dot(vectors[i+1], vectors[i]);
                result[i].x = dot(vectors[i-1], vectors[i]);
                result[i].z = dot(vectors[i+1], vectors[0]);
                result[i].w = dot(vectors[i-1], vectors[0]);
            }
        }
    }
}

__global__ void cuCrossKernel(const float3 *vectors, float3 *result, int numElements, int innerLoopSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements) {
        result[i] = vectors[i];
        if(i>1 && i < NUM_ELEMENTS - 1) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i] = cross(vectors[i-1], vectors[i]);
            }
        }
    }
}


__global__ void glmMatrixKernel(const glm::vec4 *vectors, glm::mat4 matrix, glm::vec4 *result, int numElements, int innerLoopSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements) {
        result[i] = matrix * vectors[i];
        if(i > 3) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i] = matrix * result[i];
                result[i] += matrix * vectors[i-1];
                result[i] += matrix * vectors[i-2];
                result[i] += matrix * vectors[i-3];
                result[i] += matrix * vectors[i-4];
            }
        }
    }
}
__global__ void glmDotKernel(const glm::vec4 *vectors, glm::vec4 *result, int numElements, int innerLoopSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements) {
        result[i] = vectors[i];
        if(i>1 && i < NUM_ELEMENTS - 1) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i].y = glm::dot(vectors[i+1], vectors[i]);
                result[i].x = glm::dot(vectors[i-1], vectors[i]);
                result[i].z = glm::dot(vectors[i+1], vectors[0]);
                result[i].w = glm::dot(vectors[i-1], vectors[0]);
            }
        }
    }
}

__global__ void glmCrossKernel(const glm::vec3 *vectors, glm::vec3 *result, int numElements, int innerLoopSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements) {
        result[i] = vectors[i];
        if(i>1 && i < NUM_ELEMENTS - 1) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i] = glm::cross(vectors[i-1], vectors[i]);
            }
        }
    }
}

void cpuGlmMatrixKernel(const glm::vec4 *vectors, glm::mat4 matrix, glm::vec4 *result, int numElements, int innerLoopSize) {
    for(int i=0; i<numElements; i++) {
        result[i] = matrix * vectors[i];
        if(i > 3) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i] = matrix * result[i];
                result[i] += matrix * vectors[i-1];
                result[i] += matrix * vectors[i-2];
                result[i] += matrix * vectors[i-3];
                result[i] += matrix * vectors[i-4];
            }
        }
    }
}
void cpuGlmDotKernel(const glm::vec4 *vectors, glm::vec4 *result, int numElements, int innerLoopSize) {
    for(int i=0; i<numElements; i++) {
        result[i] = vectors[i];
        if(i>1 && i < NUM_ELEMENTS - 1) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i].y = glm::dot(vectors[i+1], vectors[i]);
                result[i].x = glm::dot(vectors[i-1], vectors[i]);
                result[i].z = glm::dot(vectors[i+1], vectors[0]);
                result[i].w = glm::dot(vectors[i-1], vectors[0]);
            }
        }
    }
}

void cpuGlmCrossKernel(const glm::vec3 *vectors, glm::vec3 *result, int numElements, int innerLoopSize) {
    for(int i=0; i<numElements; i++) {
        result[i] = vectors[i];
        if(i>1 && i < NUM_ELEMENTS - 1) {
            for(int j=0; j<innerLoopSize; j++) {
                result[i] = glm::cross(vectors[i-1], vectors[i]);
            }
        }
    }
}


int main(int argc, char *argv[]) {
    glm::mat4 glmMatrix;
    glmMatrix[0] = glm::vec4(1.085f, -.15f, .72f, -0.65f);
    glmMatrix[1] = glm::vec4(.35f, -.89f, .79f, -.32f);
    glmMatrix[2] = glm::vec4(.38f, -.46f, .26f, -.83f);
    glmMatrix[3] = glm::vec4(.38f, -.80f, .90f, -.50f);

    mat4 cuMatrix = make_mat4(glmMatrix);

    glm::vec4* glmVectors = new glm::vec4[NUM_ELEMENTS];
    float4* cuVectors = new float4[NUM_ELEMENTS];
    glm::vec3* glmVectors3 = new glm::vec3[NUM_ELEMENTS];
    float3* cuVectors3 = new float3[NUM_ELEMENTS];

    for(int i=0; i<NUM_ELEMENTS; i++) {
        glmVectors[i] = glm::vec4(rand() / (float) RAND_MAX, rand() / (float) RAND_MAX, rand() / (float) RAND_MAX, rand() / (float) RAND_MAX);
        cuVectors[i] = make_float4(glmVectors[i]);
        glmVectors3[i] = glm::vec3(glmVectors[i]);
        cuVectors3[i] = make_float3(glmVectors3[i]);
    }
    glmVectors[0] = glm::vec4(1.f, 0.f, 1.f, 0.f);
    glmVectors[1] = glm::vec4(0.f, 1.f, 0.f, 1.f);
    cuVectors[0] = make_float4(glmVectors[0]);
    cuVectors[1] = make_float4(glmVectors[1]);

    glm::vec4* cpuResult = new glm::vec4[NUM_ELEMENTS];
    glm::vec3* cpuResult3 = new glm::vec3[NUM_ELEMENTS];

    size_t glmSize = NUM_ELEMENTS * sizeof(glm::vec4);
    glm::vec4* d_glmVectors;
    hce(hipMalloc(&d_glmVectors, glmSize));
    hce(hipMemcpy(d_glmVectors, glmVectors, NUM_ELEMENTS * sizeof(glm::vec4), hipMemcpyHostToDevice));
    glm::vec4* d_glmResult;
    hce(hipMalloc(&d_glmResult, glmSize));

    size_t cuSize = NUM_ELEMENTS * sizeof(float4);
    float4* d_cuVectors;
    hce(hipMalloc(&d_cuVectors, cuSize));
    hce(hipMemcpy(d_cuVectors, cuVectors, NUM_ELEMENTS * sizeof(float4), hipMemcpyHostToDevice));
    float4* d_cuResult;
    hce(hipMalloc(&d_cuResult, cuSize));


    size_t glmSize3 = NUM_ELEMENTS * sizeof(glm::vec3);
    glm::vec3* d_glmVectors3;
    hce(hipMalloc(&d_glmVectors3, glmSize3));
    hce(hipMemcpy(d_glmVectors3, glmVectors3, NUM_ELEMENTS * sizeof(glm::vec3), hipMemcpyHostToDevice));
    glm::vec3* d_glmResult3;
    hce(hipMalloc(&d_glmResult3, glmSize3));

    size_t cuSize3 = NUM_ELEMENTS * sizeof(float3);
    float3* d_cuVectors3;
    hce(hipMalloc(&d_cuVectors3, cuSize3));
    hce(hipMemcpy(d_cuVectors3, cuVectors3, NUM_ELEMENTS * sizeof(float3), hipMemcpyHostToDevice));
    float3* d_cuResult3;
    hce(hipMalloc(&d_cuResult3, cuSize3));

    int blocksPerGrid = (NUM_ELEMENTS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, THREADS_PER_BLOCK);

    //warmup
    glmMatrixKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_glmVectors, glmMatrix, d_glmResult, NUM_ELEMENTS, 10);  hce(hipDeviceSynchronize());
    glmDotKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_glmVectors, d_glmResult, NUM_ELEMENTS, 10);                hce(hipDeviceSynchronize());
    glmCrossKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_glmVectors3, d_glmResult3, NUM_ELEMENTS, 10);              hce(hipDeviceSynchronize());

    cuMatrixKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_cuVectors, cuMatrix, d_cuResult, NUM_ELEMENTS, 10);      hce(hipDeviceSynchronize());
    cuDotKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_cuVectors, d_cuResult, NUM_ELEMENTS, 10);                   hce(hipDeviceSynchronize());
    cuCrossKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_cuVectors3, d_cuResult3, NUM_ELEMENTS, 10);                 hce(hipDeviceSynchronize());

    cpuGlmMatrixKernel(glmVectors, glmMatrix, cpuResult, NUM_ELEMENTS, 10);
    cpuGlmDotKernel(glmVectors, cpuResult, NUM_ELEMENTS, 10);
    cpuGlmCrossKernel(glmVectors3, cpuResult3, NUM_ELEMENTS, 10);

    auto timeMatrix0 = std::chrono::high_resolution_clock::now();

    glmMatrixKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_glmVectors, glmMatrix, d_glmResult, NUM_ELEMENTS, 100); hce(hipDeviceSynchronize());
    auto timeMatrix1 = std::chrono::high_resolution_clock::now();

    cuMatrixKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_cuVectors, cuMatrix, d_cuResult, NUM_ELEMENTS, 100);     hce(hipDeviceSynchronize());
    auto timeMatrix2 = std::chrono::high_resolution_clock::now();

    cpuGlmMatrixKernel(glmVectors, glmMatrix, cpuResult, NUM_ELEMENTS, 100);
    auto timeMatrix3 = std::chrono::high_resolution_clock::now();

    glm::vec4* glmResult = new glm::vec4[NUM_ELEMENTS];
    hce(hipMemcpy(glmResult, d_glmResult, glmSize, hipMemcpyDeviceToHost));
    float4* cuResult = new float4[NUM_ELEMENTS];
    hce(hipMemcpy(cuResult, d_cuResult, glmSize, hipMemcpyDeviceToHost));

    hce(hipGetLastError());
    for(int i=0; i<NUM_ELEMENTS; i++) {
        if(length(cuResult[i] - make_float4(glmResult[i])) > 0.01f) {
            std::cerr << "error matrix i=" << i << std::endl;
            break;
        }
        if(glm::length(cpuResult[i] - glmResult[i]) > 0.01f) {
            std::cerr << "error matrix i=" << i << std::endl;
            break;
        }
    }

    auto timeDot0 = std::chrono::high_resolution_clock::now();

    glmDotKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_glmVectors, d_glmResult, NUM_ELEMENTS, 100);       hce(hipDeviceSynchronize());
    auto timeDot1 = std::chrono::high_resolution_clock::now();

    cuDotKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_cuVectors, d_cuResult, NUM_ELEMENTS, 100);          hce(hipDeviceSynchronize());
    auto timeDot2 = std::chrono::high_resolution_clock::now();

    cpuGlmDotKernel(glmVectors, cpuResult, NUM_ELEMENTS, 100);
    auto timeDot3 = std::chrono::high_resolution_clock::now();

    hce(hipGetLastError());

    hce(hipMemcpy(glmResult, d_glmResult, glmSize, hipMemcpyDeviceToHost));
    hce(hipMemcpy(cuResult, d_cuResult, glmSize, hipMemcpyDeviceToHost));

    for(int i=0; i<NUM_ELEMENTS; i++) {
        if(length(cuResult[i] - make_float4(glmResult[i])) > 0.0001f) {
            std::cerr << "error dot i=" << i << std::endl;
            break;
        }
        if(glm::length(cpuResult[i] - glmResult[i]) > 0.01f) {
            std::cerr << "error dot i=" << i << std::endl;
            break;
        }
    }


    auto timeCross0 = std::chrono::high_resolution_clock::now();

    glmCrossKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_glmVectors3, d_glmResult3, NUM_ELEMENTS, 100);     hce(hipDeviceSynchronize());
    auto timeCross1 = std::chrono::high_resolution_clock::now();

    cuCrossKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_cuVectors3, d_cuResult3, NUM_ELEMENTS, 100);        hce(hipDeviceSynchronize());
    auto timeCross2 = std::chrono::high_resolution_clock::now();

    cpuGlmCrossKernel(glmVectors3, cpuResult3, NUM_ELEMENTS, 100);
    auto timeCross3 = std::chrono::high_resolution_clock::now();

    hce(hipGetLastError());

    glm::vec3* glmResult3 = new glm::vec3[NUM_ELEMENTS];
    hce(hipMemcpy(glmResult3, d_glmResult3, glmSize3, hipMemcpyDeviceToHost));
    float3* cuResult3 = new float3[NUM_ELEMENTS];
    hce(hipMemcpy(cuResult3, d_cuResult3, cuSize3, hipMemcpyDeviceToHost));

    for(int i=0; i<NUM_ELEMENTS; i++) {
        if(length(cuResult3[i] - make_float3(glmResult3[i])) > 0.0001f) {
            std::cerr << "error cross i=" << i << std::endl;
            break;
        }
        if(glm::length(cpuResult3[i] - glmResult3[i]) > 0.01f) {
            std::cerr << "error cross i=" << i << std::endl;
            break;
        }
    }

    std::cout << "time for cpu glm (matrix):          " << std::chrono::duration_cast<std::chrono::milliseconds>(timeMatrix3 -  timeMatrix2).count() << " milliseconds" << std::endl;
    std::cout << "time for cuda glm (matrix):         " << std::chrono::duration_cast<std::chrono::milliseconds>(timeMatrix1 -  timeMatrix0).count() << " milliseconds" << std::endl;
    std::cout << "time for cuda helper math (matrix): " << std::chrono::duration_cast<std::chrono::milliseconds>(timeMatrix2 -  timeMatrix1).count() << " milliseconds" << std::endl;

    std::cout << "time for cpu glm (dot):             " << std::chrono::duration_cast<std::chrono::milliseconds>(timeDot3 -     timeDot2).count() << " milliseconds" << std::endl;
    std::cout << "time for cuda glm (dot):            " << std::chrono::duration_cast<std::chrono::milliseconds>(timeDot1 -     timeDot0).count() << " milliseconds" << std::endl;
    std::cout << "time for cuda helper math (dot):    " << std::chrono::duration_cast<std::chrono::milliseconds>(timeDot2 -     timeDot1).count() << " milliseconds" << std::endl;

    std::cout << "time for cpu glm (cross):           " << std::chrono::duration_cast<std::chrono::milliseconds>(timeCross3 -   timeCross2).count() << " milliseconds" << std::endl;
    std::cout << "time for cuda glm (cross):          " << std::chrono::duration_cast<std::chrono::milliseconds>(timeCross1 -   timeCross0).count() << " milliseconds" << std::endl;
    std::cout << "time for cuda helper math (cross):  " << std::chrono::duration_cast<std::chrono::milliseconds>(timeCross2 -   timeCross1).count() << " milliseconds" << std::endl;

    delete[] glmVectors;
    delete[] cuVectors;
    delete[] cpuResult;
    delete[] glmResult;
    delete[] cuResult;

    hipFree(d_glmVectors);
    hipFree(d_glmResult);
    hipFree(d_cuVectors);
    hipFree(d_cuResult);

    return 0;
}
